#include "hip/hip_runtime.h"
#include "gaussian.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

#include <>

namespace CUDA_Helper
{
	__global__ void GaussianKernal(int maxCount, int width, int height, uchar4* out_image, uchar4* in_image, float* kernel, int const kernelHalfRadius)
	{
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index >= maxCount) return;

		int id_x = index % width;
		int id_y = (index - id_x) / width;

		int kernel_radius = kernelHalfRadius * 2 + 1;

		float4 pixel_sum = { 0.f, 0.f, 0.f, 0.f };
		float weight_sum = 0.f;

		for (int ky = -kernelHalfRadius; ky <= kernelHalfRadius; ky++) 
		{
			for (int kx = -kernelHalfRadius; kx <= kernelHalfRadius; kx++) 
			{
				int x = id_x + kx;
				int y = id_y + ky;
				// Boundary check
				if (y >= 0 && y < height && x >= 0 && x < width) 
				{
					uchar4 const& pixel = in_image[y * width + x];
					float weight = kernel[(ky + kernelHalfRadius) * kernel_radius + kx + kernelHalfRadius];

					pixel_sum.x += static_cast<float>(pixel.x) * weight;
					pixel_sum.y += static_cast<float>(pixel.y) * weight;
					pixel_sum.z += static_cast<float>(pixel.z) * weight;
					pixel_sum.w += static_cast<float>(pixel.w) * weight;

					weight_sum += weight;
				}
			}
		}

		uchar4 result;
		result.x = static_cast<unsigned char>(pixel_sum.x / weight_sum);
		result.y = static_cast<unsigned char>(pixel_sum.y / weight_sum);
		result.z = static_cast<unsigned char>(pixel_sum.z / weight_sum);
		result.w = static_cast<unsigned char>(pixel_sum.w / weight_sum);
		out_image[index] = result;
	}

	void GaussianBlur(void* const out_image, void const* const input_image, int const& width, int const& height, int const& kernelHalfRadius, float* kernel)
	{
		int max_count = width * height;
		int num_block = (max_count + 127) / 128;
		
		uchar4 const* in = reinterpret_cast<uchar4 const*>(input_image);
		uchar4* out = reinterpret_cast<uchar4*>(out_image);

		thrust::device_vector<uchar4> thrust_dev_in(in, in + max_count);
		thrust::device_vector<uchar4> thrust_dev_out(out, out + max_count);

		uint16_t kernel_radius = (kernelHalfRadius << 1) + 1;
		uint16_t kernel_size = kernel_radius * kernel_radius;
		thrust::device_vector<float> thrust_dev_kernel(kernel, kernel + kernel_size);

		GaussianKernal << <num_block, 128 >> > (max_count, width, height, 
			thrust_dev_out.data().get(), 
			thrust_dev_in.data().get(), 
			thrust_dev_kernel.data().get(), 
			kernelHalfRadius);

		hipMemcpy(out_image, thrust_dev_out.data().get(), max_count * sizeof(uchar4), hipMemcpyDeviceToHost);
	}
}